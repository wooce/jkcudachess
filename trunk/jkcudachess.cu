/*
 * Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and
 * international Copyright laws.  Users and possessors of this source code
 * are hereby granted a nonexclusive, royalty-free license to use this code
 * in individual and commercial software.
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
 * OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
 * OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
 * OR PERFORMANCE OF THIS SOURCE CODE.
 *
 * U.S. Government End Users.   This source code is a "commercial item" as
 * that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
 * "commercial computer  software"  and "commercial computer software
 * documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
 * and is provided to the U.S. Government only as a commercial end item.
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
 * source code with only those rights set forth herein.
 *
 * Any use of this source code in individual and commercial software must
 * include, in the user documentation and internal comments to the code,
 * the above Disclaimer and U.S. Government End Users Notice.
 */

/* jkcudachess project which demonstrates the basics on how to setup a project 
* example application.
* Host code.
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
#include <cutil.h>

// includes, kernels
#include <jkcudachess_kernel.cu>

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

extern "C"
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    runTest( argc, argv);

    CUT_EXIT(argc, argv);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{

    CUT_DEVICE_INIT(argc, argv);

    unsigned int timer = 0;
    CUT_SAFE_CALL( cutCreateTimer( &timer));
    CUT_SAFE_CALL( cutStartTimer( timer));

    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof( float) * num_threads;

    // allocate host memory
    float* h_idata = (float*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float* d_idata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_idata, mem_size));
    // copy host memory to device
    CUDA_SAFE_CALL( hipMemcpy( d_idata, h_idata, mem_size,
                                hipMemcpyHostToDevice) );

    // allocate device memory for result
    float* d_odata;
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_odata, mem_size));

    // setup execution parameters
    dim3  grid( 1, 1, 1);
    dim3  threads( num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>( d_idata, d_odata);

    // check if kernel execution generated and error
    CUT_CHECK_ERROR("Kernel execution failed");

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    // copy result from device to host
    CUDA_SAFE_CALL( hipMemcpy( h_odata, d_odata, sizeof( float) * num_threads,
                                hipMemcpyDeviceToHost) );

    CUT_SAFE_CALL( cutStopTimer( timer));
    printf( "Processing time: %f (ms)\n", cutGetTimerValue( timer));
    CUT_SAFE_CALL( cutDeleteTimer( timer));

    // compute reference solution
    float* reference = (float*) malloc( mem_size);
    computeGold( reference, h_idata, num_threads);

    // check result
    if( cutCheckCmdLineFlag( argc, (const char**) argv, "regression")) 
    {
        // write file for regression test
        CUT_SAFE_CALL( cutWriteFilef( "./data/regression.dat",
                                      h_odata, num_threads, 0.0));
    }
    else 
    {
        // custom output handling when no regression test running
        // in this case check if the result is equivalent to the expected soluion
        CUTBoolean res = cutComparef( reference, h_odata, num_threads);
        printf( "jkTest %s\n", (1 == res) ? "PASSED" : "FAILED");
    }

    // cleanup memory
    free( h_idata);
    free( h_odata);
    free( reference);
    CUDA_SAFE_CALL(hipFree(d_idata));
    CUDA_SAFE_CALL(hipFree(d_odata));
}

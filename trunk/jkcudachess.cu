#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////
// �����JBinghewusi.cpp                                                                                 //
// *******************************************************************************************************//
// ����H�ѳq�Τ���----�L�e���|�A����m����H�ѳq�Τ�����ĳ�n(Universal Chinese Chess Protocol�A²��ucci) //
// �@�̡J �S �w �x                                                                                        //
// ���J �����l���Ǭ�s�|                                                                            //
// �l�c�J fan_de_jun@sina.com.cn                                                                          //
//  QQ �J 83021504                                                                                        //
// *******************************************************************************************************//
// �\��J                                                                                                 //
// 1. ����x���ε{�����J�f�I                                                                              //
// 2. �z�Lucci��ĳ�P�ɭ��{�������i��q�T                                                                  //
////////////////////////////////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// includes, project
#include <cutil.h>
#include <cutil_inline.h>
// includes, kernels
#include <jkcudachess_kernel.cu>
// includes, Binghewusi
#include <string.h>
#include "ucci.h"
#include "FenBoard.h"
#include "Search.h"

int main(int argc, char* argv[])
{
    //call_vecAdd();
    int n;
    const char *BoolValue[2] = { "false", "true" };
    const char *ChessStyle[3] = { "solid", "normal", "risky" };
    char *BackSlashPtr;
    char BookFile[1024];
    UcciCommEnum IdleComm;
    UcciCommStruct Command;
    int ThisMove;

    printf("*******************************�L�e���| V0.60*********************************\n");
    printf("** �@�̡J�S�w�x                                                             **\n");
    printf("** ����m����H�ѳq�Τ�����ĳ�n(Universal Chinese Chess Protocol�A²��UCCI) **\n");
    printf("** �����i�H�Ρ��H����ElephantBoard�ɭ��{��(�@�̡J����)�[���C                **\n");
    printf("** �w��ϥΡ��L�e���| V0.60������H�Ѥ���                                   **\n");	
    printf("******************************************************************************\n");
    printf("����Jucci���O......\n");

    // ��������"ucci"���O
    if(BootLine() == UCCI_COMM_UCCI)
    {
        // �M������Ҧb���ؿ�argv[0]�A�åB��"BOOK.DAT"�q�{���ʬ٪��}���w�}���w
        BackSlashPtr = strrchr(argv[0], '\\');
        if (BackSlashPtr == 0) 
            strcpy(BookFile, "BOOK.DAT");
        else
        {
            strncpy(BookFile, argv[0], BackSlashPtr + 1 - argv[0]);
            strcpy(BookFile + (BackSlashPtr + 1 - argv[0]), "BOOK.DAT");
        }

        // �ե�CSearch���A�c�y��ƪ�l�Ƥ@�Ǭ����Ѽ�
        //a.��l�Ƶ۪k�w���ͼƲ�
        //b.��l��Hash��A���o21+1=22��Hash��A64M
        //c.�M�ž��v�ҵo��
        CSearch ThisSearch;

        // ��ܤ������W�١B�����B�@�̩M�ϥΪ�
        printf("\n");
        printf("id name ����H�Ѥ����L�e���|V0.60jk��\n");
        fflush(stdout);
        printf("id copyright ���v�Ҧ�(C) 2005-2008\n");
        fflush(stdout);
        printf("id author �S�w�x(�����l���Ǭ�s�|)\n");
        fflush(stdout);
        printf("id user CUDA\n\n");
        fflush(stdout);

        // ��ܤ���ucci���O���^�X�T���A��ܤ����Ҥ�����ﶵ
        // option batch %d
        printf("option batch type check default %s\n", BoolValue[ThisSearch.bBatch]);
        fflush(stdout);

        // option debug ��������X�ԲӪ��j���T���A�ëD�u�����ոռҦ��C
        printf("option debug type check default %s\n", BoolValue[ThisSearch.Debug]);
        fflush(stdout);

        // ���w�}���w��󪺦W�١A�i���w�h�Ӷ}���w���A�Τ�����;���j�}�A�p���������ϥζ}���w�A�i�H��ȳ]����
        ThisSearch.bUseOpeningBook = ThisSearch.m_Hash.LoadBook(BookFile);
        if(ThisSearch.bUseOpeningBook)
            printf("option bookfiles type string default %s\n", BookFile);
        else
            printf("option bookfiles type string default %s\n", 0);
        fflush(stdout);

        // �ݧ��w�W��
        printf("option egtbpaths type string default null\n");
        fflush(stdout);

        // ���Hash���j�p
        printf("option hashsize type spin default %d MB\n", ThisSearch.m_Hash.nHashSize*2*sizeof(CHashRecord)/1024/1024);
        fflush(stdout);

        // �������u�{��
        printf("option threads type spin default %d\n", 0);
        fflush(stdout);

        // �����F��۵M���۪��b�^�X��
        printf("option drawmoves type spin default %d\n", ThisSearch.NaturalBouts);
        fflush(stdout);

        // �ѳW
        printf("option repetition type spin default %d 1999�~���m����H���v�ɳW�h�n\n", UCCI_REPET_CHINESERULE);
        fflush(stdout);

        // �ŵ۵���O�_���}
        printf("option pruning type check %d\n", ThisSearch);
        fflush(stdout);

        // ���Ȩ�ƪ��ϥα��p
        printf("option knowledge type check %d\n", ThisSearch);
        fflush(stdout);

        // ���w��ܩʨt�ơA�q�`��0,1,2,3�|�ӯŧO�C�����Ȩ�ƥ[��@�w�d�򤺪��H���ơA�������C�����X���ۦP���ѡC
        printf("option selectivity type spin min 0 max 3 default %d\n", ThisSearch.nSelectivity);
        fflush(stdout);

        // ���w�U�Ѫ�����A�q�`��solid(�O�u)�Bnormal(����)�Mrisky(�_�i)�T��
        printf("option style type combo var solid var normal var risky default %s\n", ChessStyle[ThisSearch.nStyle]);
        fflush(stdout);		

        // copyprotection ��ܪ��v�ˬd�T��(���b�ˬd�A���v�T�����T�Ϊ��v�T�����~)�C 
        printf("copyprotection ok\n\n");
        fflush(stdout);

        // ucciok �o�Oucci���O���̫�@���^�X�T���A��ܤ����w�g�i�J��UCCI��ĳ�q�T�����A�C
        printf("ucciok\n\n");
        fflush(stdout);


        // �]�w�зǿ�X�M��l����
        ThisSearch.OutFile = stdout;	// �зǿ�X
        ThisSearch.fen.FenToBoard(Board, Piece, ThisSearch.Player, ThisSearch.nNonCapNum, ThisSearch.nCurrentStep, "rnbakabnr/9/1c5c1/p1p1p1p1p/9/9/P1P1P1P1P/1C5C1/9/RNBAKABNR r - - 0 1");
        ThisSearch.InitBitBoard(ThisSearch.Player, ThisSearch.nCurrentStep);
        printf("position fen rnbakabnr/9/1c5c1/p1p1p1p1p/9/9/P1P1P1P1P/1C5C1/9/RNBAKABNR r - - 0 1\n\n");
        fflush(stdout);


        // �}�l��������UCCI�R�O
        do 
        {
            IdleComm = IdleLine(Command, ThisSearch.Debug);
            switch (IdleComm) 
            {
                // isready �˴������O�_�B�_�N�����A�A��^�X�T���`�Oreadyok�A�ӫ��O�ȶȥΨ��˴������������O�����w�R�ϡ��O�_�ॿ�`�e�ǫ��O�C
                // readyok ��������B�_�N�����A(�Y�i�������O�����A)�A���ޤ����B�_�Ŷ����A�٬O��Ҫ��A�C
            case UCCI_COMM_ISREADY:
                printf("readyok\n");
                fflush(stdout);
                break;

                // stop ���_��������ҡA�j�s�X�ۡC�Z�x��ҨS���R���ɡA�N�θӫ��O�Ӥ����ҡA�M�᭫�s��J�����C
            case UCCI_COMM_STOP:
                ThisSearch.bStopThinking = 1;
                //printf("nobestmove\n");
                printf("score 0\n");
                fflush(stdout);
                break;

                // position fen �]�m�����m�ѽL���������A��fen�ӫ��wFEN�榡��Amoves�Z���򪺬O�H�Z���L���۪k
            case UCCI_COMM_POSITION:
                // �N�ɭ��ǨӪ�Fen����Ƭ��ѧ��T��
                ThisSearch.fen.FenToBoard(Board, Piece, ThisSearch.Player, ThisSearch.nNonCapNum, ThisSearch.nCurrentStep, Command.Position.szFenStr);
                ThisSearch.InitBitBoard(ThisSearch.Player, ThisSearch.nCurrentStep);

                // �N���������e�A�D�n�O���F��s�۪k�O���A�Τ_�`���˴��C
                for(n=0; n<Command.Position.nMoveNum; n++)
                {
                    ThisMove = Move(Command.Position.lpdwCoordList[n]);
                    if( !ThisMove )
                        break;

                    ThisSearch.MovePiece( ThisMove );
                    ThisSearch.StepRecords[ThisSearch.nCurrentStep-1] |= ThisSearch.Checking(ThisSearch.Player) << 24;
                }

                ThisSearch.nBanMoveNum = 0;
                break;

                // banmoves ����e�����]�m�T��A�H�ѨM�����L�k�B�z���������D�C��X�{���������ɡA�Ѥ�i�H�ޱ��ɭ��V�����o�X�T����O�C
            case UCCI_COMM_BANMOVES:
                ThisSearch.nBanMoveNum = Command.BanMoves.nMoveNum;
                for(n=0; n<Command.BanMoves.nMoveNum; n++)
                    ThisSearch.BanMoveList[n] = Move(Command.BanMoves.lpdwCoordList[n]);
                break;

                // setoption �]�m�����U�ذѼ�
            case UCCI_COMM_SETOPTION:
                switch(Command.Option.uoType) 
                {
                    // setoption batch %d
                case UCCI_OPTION_BATCH:
                    ThisSearch.bBatch = (Command.Option.Value.bCheck == TRUE);
                    printf("option batch type check default %s\n", BoolValue[ThisSearch.bBatch]);
                    fflush(stdout);
                    break;

                    // setoption debug %d ��������X�ԲӪ��j���T���A�ëD�u�����ոռҦ��C
                case UCCI_OPTION_DEBUG:
                    ThisSearch.Debug = (Command.Option.Value.bCheck == TRUE);
                    printf("option debug type check default %s\n", BoolValue[ThisSearch.Debug]);
                    fflush(stdout);
                    break;

                    // setoption bookfiles %s  ���w�}���w��󪺦W�١A�i���w�h�Ӷ}���w���A�Τ�����;���j�}�A�p���������ϥζ}���w�A�i�H��ȳ]����
                case UCCI_OPTION_BOOKFILES:
                    strcpy(BookFile, Command.Option.Value.szString);
                    printf("option bookfiles type string default %s\n", BookFile);
                    fflush(stdout);
                    break;

                    // setoption egtbpaths %s  ���w�ݧ��w��󪺦W�١A�i���w�h�Ӵݧ��w���|�A�Τ�����;���j�}�A�p���������ϥδݧ��w�A�i�H��ȳ]����
                    //case e_OptionEgtbPaths:
                    // �����ثe������}���w
                    //printf("option egtbpaths type string default null\n");
                    //fflush(stdout);
                    //break;

                    // setoption hashsize %d  �HMB�����W�wHash���j�p�A-1����������۰ʤ��oHash��C1��1024MB
                    // �H���ɭ�����Bug�A�C���]�m�����ɡA�o�өR�O���b�}���w���e��
                case UCCI_OPTION_HASHSIZE:
                    // -1MB(�۰�), 0MB(�۰�), 1MB(16), 2MB(17), 4MB(18), 8MB(19), 16MB(20), 32MB(21), 64MB(22), 128MB(23), 256MB(24), 512MB(25), 1024MB(26)
                    if( Command.Option.Value.nSpin <= 0)
                        n = 22;		// �ʬٱ��p�U�A�����۰ʤ��o(1<<22)*16=64MB�A���P�¨�U��A����U�@�b�C
                    else
                    {
                        n = 15;											// 0.5 MB = 512 KB �H�������
                        while( Command.Option.Value.nSpin > 0 )
                        {
                            Command.Option.Value.nSpin >>= 1;			// �C�����H2�A���쬰0
                            n ++;
                        }
                    }								

                    // ���[�J���s�˴����s�A�����۰ʤ��o�ɡAHash��j�p���i�Τ��s��1/2�C
                    ThisSearch.m_Hash.DeleteHashTable();					// �����ϥ�delete���M���ª�Hash��
                    ThisSearch.m_Hash.NewHashTable(n > 26 ? 26 : n, 12);	// ���������o�s��Hash��
                    printf("option hashsize type spin default %d MB\n", ThisSearch.m_Hash.nHashSize*2*sizeof(CHashRecord)/1024/1024);	// ��ܹ�ڤ��o��Hash��j�p�A���JMB
                    fflush(stdout);

                    ThisSearch.m_Hash.ClearHashTable();
                    ThisSearch.bUseOpeningBook = ThisSearch.m_Hash.LoadBook(BookFile);
                    break;

                    // setoption threads %d	      �������u�{�ơA���h�B�z���æ�B��A��
                case UCCI_OPTION_THREADS:
                    // ThisSearch.nThreads = Command.Option.Value.Spin;		// 0(auto),1,2,4,8,16,32
                    printf("option drawmoves type spin default %d\n", 0);
                    fflush(stdout);
                    break;

                    // setoption drawmoves %d	  �F��۵M���۪��^�X��:50,60,70,80,90,100�A�H���w�g�۰���Ƭ��b�^�X��
                case UCCI_OPTION_DRAWMOVES:
                    ThisSearch.NaturalBouts = Command.Option.Value.nSpin;
                    printf("option drawmoves type spin default %d\n", ThisSearch.NaturalBouts);
                    fflush(stdout);
                    break;

                    // setoption repetition %d	  �B�z�`�����ѳW�A�ثe�u���������H�ѴѳW1999��
                case UCCI_OPTION_REPETITION:
                    // ThisSearch.nRepetitionStyle = Command.Option.Value.Repetition;
                    // e_RepetitionAlwaysDraw  ���ܧ@�M
                    // e_RepetitionCheckBan    �T����N
                    // e_RepetitionAsianRule   �Ȭw�W�h
                    // e_RepetitionChineseRule ����W�h�]�ʬ١^
                    printf("option repetition type spin default %d", UCCI_REPET_CHINESERULE);
                    printf("  ���L�e���|�������ثe���1999�~���m����H���v�ɳW�h�n\n");
                    fflush(stdout);
                    break;

                    // setoption pruning %d�A���ŵۦV�e���š��O�_���}
                case UCCI_OPTION_PRUNING:
                    ThisSearch.bPruning = Command.Option.Value.ugGrade;
                    printf("option pruning type check %d\n", ThisSearch);
                    fflush(stdout);
                    break;

                    // setoption knowledge %d�A���Ȩ�ƪ��ϥ�
                case UCCI_OPTION_KNOWLEDGE:
                    ThisSearch.bKnowledge = Command.Option.Value.ugGrade;
                    printf("option knowledge type check %d\n", ThisSearch);
                    fflush(stdout);
                    break;

                    // setoption selectivity %d  ���w��ܩʨt�ơA�q�`��0,1,2,3�|�ӯŧO
                case UCCI_OPTION_SELECTIVITY:
                    switch (Command.Option.Value.ugGrade)
                    {
                    case UCCI_GRADE_NONE:
                        ThisSearch.SelectMask = 0;
                        break;
                    case UCCI_GRADE_SMALL:
                        ThisSearch.SelectMask = 1;
                        break;
                    case UCCI_GRADE_MEDIUM:
                        ThisSearch.SelectMask = 3;
                        break;
                    case UCCI_GRADE_LARGE:
                        ThisSearch.SelectMask = 7;
                        break;
                    default:
                        ThisSearch.SelectMask = 0;
                        break;
                    }
                    printf("option selectivity type spin min 0 max 3 default %d\n", ThisSearch.SelectMask);
                    fflush(stdout);
                    break;

                    // setoption style %d  ���w�U�Ѫ�����A�q�`��solid(�O�u)�Bnormal(����)�Mrisky(�_�i)�T��
                case UCCI_OPTION_STYLE:
                    ThisSearch.nStyle = Command.Option.Value.usStyle;
                    printf("option style type combo var solid var normal var risky default %s\n", ChessStyle[Command.Option.Value.usStyle]);
                    fflush(stdout);
                    break;						

                    // setoption loadbook  UCCI�ɭ�ElephantBoard�b�C���s�شѧ��ɳ��|�o�e�o�����O
                case UCCI_OPTION_LOADBOOK:
                    ThisSearch.m_Hash.ClearHashTable();
                    ThisSearch.bUseOpeningBook = ThisSearch.m_Hash.LoadBook(BookFile);

                    if(ThisSearch.bUseOpeningBook)
                        printf("option loadbook succeed. %s\n", BookFile);		// ���\
                    else
                        printf("option loadbook failed! %s\n", "Not found file BOOK.DAT");				// �S���}���w
                    fflush(stdout);
                    printf("\n\n");
                    fflush(stdout);
                    break;

                default:
                    break;
                }
                break;

                // Prepare timer strategy according to "go depth %d" or "go ponder depth %d" command
            case UCCI_COMM_GO:
            case UCCI_COMM_GOPONDER:
                switch (Command.Search.utMode)
                {
                    // �T�w�`��
                case UCCI_TIME_DEPTH:
                    ThisSearch.Ponder = 2;
                    ThisSearch.MainSearch(Command.Search.DepthTime.nDepth);
                    break;

                    // �ɬq�s�J ���o�ɶ� = �Ѿl�ɶ� / �n�����B��
                case UCCI_TIME_MOVE:							
                    ThisSearch.Ponder = (IdleComm == UCCI_COMM_GOPONDER ? 1 : 0);
                    printf("%d\n", Command.Search.TimeMode.nMovesToGo);
                    ThisSearch.MainSearch(127, Command.Search.DepthTime.nTime * 1000 / Command.Search.TimeMode.nMovesToGo, Command.Search.DepthTime.nTime * 1000);
                    break;

                    // �[�ɻs�J ���o�ɶ� = �C�B�W�[���ɶ� + �Ѿl�ɶ� / 20 (�Y���]�ѧ��|�b20�B������)
                case UCCI_TIME_INC:
                    ThisSearch.Ponder = (IdleComm == UCCI_COMM_GOPONDER ? 1 : 0);
                    ThisSearch.MainSearch(127, (Command.Search.DepthTime.nTime + Command.Search.TimeMode.nIncrement * 20) * 1000 / 20, Command.Search.DepthTime.nTime * 1000);
                    break;

                default:
                    break;
                }
                break;
            }
        } while (IdleComm != UCCI_COMM_QUIT);

        printf("bye\n");
        fflush(stdout);
    }

    return 0;
}

//�I�s�H���üư}�C�[�k����
void call_vecAdd()
{
    //printf("movegen_calls=%d\n",movegen_calls);
    /*
    printf("test start");

    unsigned int num_threads = 4088;
    int  MAX_BLOCKTHREAD=512;
    unsigned int mem_size = sizeof( float) * num_threads;

    // allocate host memory
    float* h_idata1 = (float*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
    h_idata1[i] = (float) (rand()%100);
    }
    float* h_idata2 = (float*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
    h_idata2[i] = (float) (rand()%100);
    }

    // allocate device memory
    float* d_idata1;
    cutilSafeCall( hipMalloc( (void**) &d_idata1, mem_size));
    // copy host memory to device
    cutilSafeCall( hipMemcpy( d_idata1, h_idata1, mem_size, hipMemcpyHostToDevice) );
    float* d_idata2;
    cutilSafeCall( hipMalloc( (void**) &d_idata2, mem_size));
    // copy host memory to device
    cutilSafeCall( hipMemcpy( d_idata2, h_idata2, mem_size, hipMemcpyHostToDevice) );

    // allocate device memory for result
    float* d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));

    // execute the kernel
    dim3  grid( (( num_threads -1 ) / MAX_BLOCKTHREAD + 1) , 1);
    dim3  threads( MAX_BLOCKTHREAD , 1);
    vecAdd<<< grid, threads, mem_size >>>( d_idata1, d_idata2, d_odata);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    // copy result from device to host
    cutilSafeCall( hipMemcpy( h_odata, d_odata, sizeof( float) * num_threads, hipMemcpyDeviceToHost) );

    //�L�X���G
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
    printf("%f + %f = %f \n",h_idata1[i],h_idata2[i],h_odata[i]);
    }

    system("pause");
    */
}

#define numOfThreads 180

texture<unsigned char, 1, hipReadModeElementType> texture_KingMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_xRookMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_yRookMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_xCannonMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_yCannonMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_KnightMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_BishopMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_GuardMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_PawnMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_xRookCapMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_yRookCapMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_xCannonCapMoves;
texture<unsigned char, 1, hipReadModeElementType> texture_yCannonCapMoves;
//�w�i�}�B��
unsigned char * cuda_KingMoves;
unsigned char * cuda_xRookMoves;
unsigned char * cuda_yRookMoves;
unsigned char * cuda_xCannonMoves;
unsigned char * cuda_yCannonMoves;
unsigned char * cuda_KnightMoves;
unsigned char * cuda_BishopMoves;
unsigned char * cuda_GuardMoves;
unsigned char * cuda_PawnMoves;
unsigned char * cuda_xRookCapMoves;
unsigned char * cuda_yRookCapMoves;
unsigned char * cuda_xCannonCapMoves;
unsigned char * cuda_yCannonCapMoves;
//�t�mGPU�s�񵲪G���O����
unsigned int* cuda_move;

__constant__ unsigned int cuda_xBitBoard[16];//16*4=64
__constant__ unsigned int cuda_yBitBoard[16];//16*4=64
__constant__ int cuda_Board[256];//256*4=1024
__constant__ int cuda_Piece[48];//48*4=192
__constant__ char cuda_nHorseLegTab[512] = {// ���L�W�q��
    0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,-16,  0,-16,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0, -1,  0,  0,  0,  1,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0, -1,  0,  0,  0,  1,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0, 16,  0, 16,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0,
    0,  0,  0,  0,  0,  0,  0
};
//�ƻsPreMoveGen�i�}�����k��GPU���O�����
void copyPreMoveToGPU(unsigned char host_KingMoves[256][8],unsigned char host_xRookMoves[12][512][12],unsigned char host_yRookMoves[13][1024][12],unsigned char host_xCannonMoves[12][512][12],unsigned char host_yCannonMoves[13][1024][12],unsigned char host_KnightMoves[256][12],unsigned char host_BishopMoves[256][8],unsigned char host_GuardMoves[256][8],unsigned char host_PawnMoves[2][256][4],unsigned char host_xRookCapMoves[12][512][4],unsigned char host_yRookCapMoves[13][1024][4],unsigned char host_xCannonCapMoves[12][512][4],unsigned char host_yCannonCapMoves[13][1024][4])
{
    int MEMSIZE_KingMoves=2048;
    int MEMSIZE_xRookMoves=73728;
    int MEMSIZE_yRookMoves=159744;
    int MEMSIZE_xCannonMoves=73728;
    int MEMSIZE_yCannonMoves=159744;
    int MEMSIZE_KnightMoves=3072;
    int MEMSIZE_BishopMoves=2048;
    int MEMSIZE_GuardMoves=2048;
    int MEMSIZE_PawnMoves=2048;
    int MEMSIZE_xRookCapMoves=24576;
    int MEMSIZE_yRookCapMoves=53248;
    int MEMSIZE_xCannonCapMoves=24576;
    int MEMSIZE_yCannonCapMoves=53248;

    //hipChannelFormatDesc channelDesc = hipCreateChannelDesc<unsigned char>();
    //const hipExtent KingMoves_volumeSize = make_hipExtent(256, 8, 0);
    //hipArray *cuda_KingMovesArray;
    //cutilSafeCall(hipMalloc3DArray(&cuda_KingMovesArray, &channelDesc, KingMoves_volumeSize) );
    //// copy data to 3D array
    //hipMemcpy3DParms KingMoves_copyParams = {0};
    //KingMoves_copyParams.srcPtr   = make_hipPitchedPtr((void*)host_KingMoves, KingMoves_volumeSize.width*sizeof(unsigned char), KingMoves_volumeSize.width, KingMoves_volumeSize.height);
    //KingMoves_copyParams.dstArray = cuda_KingMovesArray;
    //KingMoves_copyParams.extent   = KingMoves_volumeSize;
    //KingMoves_copyParams.kind     = hipMemcpyHostToDevice;
    //cutilSafeCall(hipMemcpy3D(&KingMoves_copyParams));
    //cutilSafeCall(hipBindTextureToArray(texture_KingMoves,cuda_KingMovesArray,channelDesc));

    cutilSafeCall(hipMalloc( (void**) &cuda_KingMoves, MEMSIZE_KingMoves));
    cutilSafeCall(hipMemcpy( cuda_KingMoves, host_KingMoves, MEMSIZE_KingMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_KingMoves,cuda_KingMoves,MEMSIZE_KingMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_xRookMoves, MEMSIZE_xRookMoves));
    cutilSafeCall(hipMemcpy( cuda_xRookMoves, host_xRookMoves, MEMSIZE_xRookMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_xRookMoves,cuda_xRookMoves,MEMSIZE_xRookMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_yRookMoves, MEMSIZE_yRookMoves));
    cutilSafeCall(hipMemcpy( cuda_yRookMoves, host_yRookMoves, MEMSIZE_yRookMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_yRookMoves,cuda_yRookMoves,MEMSIZE_yRookMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_xCannonMoves, MEMSIZE_xCannonMoves));
    cutilSafeCall(hipMemcpy( cuda_xCannonMoves, host_xCannonMoves, MEMSIZE_xCannonMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_xCannonMoves,cuda_xCannonMoves,MEMSIZE_xCannonMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_yCannonMoves, MEMSIZE_yCannonMoves));
    cutilSafeCall(hipMemcpy( cuda_yCannonMoves, host_yCannonMoves, MEMSIZE_yCannonMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_yCannonMoves,cuda_yCannonMoves,MEMSIZE_yCannonMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_KnightMoves, MEMSIZE_KnightMoves));
    cutilSafeCall(hipMemcpy( cuda_KnightMoves, host_KnightMoves, MEMSIZE_KnightMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_KnightMoves,cuda_KnightMoves,MEMSIZE_KnightMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_BishopMoves, MEMSIZE_BishopMoves));
    cutilSafeCall(hipMemcpy( cuda_BishopMoves, host_BishopMoves, MEMSIZE_BishopMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_BishopMoves,cuda_BishopMoves,MEMSIZE_BishopMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_GuardMoves, MEMSIZE_GuardMoves));
    cutilSafeCall(hipMemcpy( cuda_GuardMoves, host_GuardMoves, MEMSIZE_GuardMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_GuardMoves,cuda_GuardMoves,MEMSIZE_GuardMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_PawnMoves, MEMSIZE_PawnMoves));
    cutilSafeCall(hipMemcpy( cuda_PawnMoves, host_PawnMoves, MEMSIZE_PawnMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_PawnMoves,cuda_PawnMoves,MEMSIZE_PawnMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_xRookCapMoves, MEMSIZE_xRookCapMoves));
    cutilSafeCall(hipMemcpy( cuda_xRookCapMoves, host_xRookCapMoves, MEMSIZE_xRookCapMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_xRookCapMoves,cuda_xRookCapMoves,MEMSIZE_xRookCapMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_yRookCapMoves, MEMSIZE_yRookCapMoves));
    cutilSafeCall(hipMemcpy( cuda_yRookCapMoves, host_yRookCapMoves, MEMSIZE_yRookCapMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_yRookCapMoves,cuda_yRookCapMoves,MEMSIZE_yRookCapMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_xCannonCapMoves, MEMSIZE_xCannonCapMoves));
    cutilSafeCall(hipMemcpy( cuda_xCannonCapMoves, host_xCannonCapMoves, MEMSIZE_xCannonCapMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_xCannonCapMoves,cuda_xCannonCapMoves,MEMSIZE_xCannonCapMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_yCannonCapMoves, MEMSIZE_yCannonCapMoves));
    cutilSafeCall(hipMemcpy( cuda_yCannonCapMoves, host_yCannonCapMoves, MEMSIZE_yCannonCapMoves, hipMemcpyHostToDevice));
    cutilSafeCall(hipBindTexture(0, texture_yCannonCapMoves,cuda_yCannonCapMoves,MEMSIZE_yCannonCapMoves));

    cutilSafeCall(hipMalloc( (void**) &cuda_move, 4*numOfThreads));
}
//cuda MoveGen�D��
#define WRITE_2_MOVE if(pMove&&nSrc&&nDst&&!cuda_Board[nDst]){move=(nSrc<<8)|nDst;}else{move=0;}
__global__ void cudaMoveGen_warp(const unsigned int k,unsigned int* cuda_move)
{    
    //int tid=blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int  move, nSrc, nDst, x, y, nChess;
    unsigned char pMove;
    //�N��**************************************************************************
    if(blockIdx.x==0)
    {
        if(threadIdx.x<8)
        {
            nChess=k;
            nSrc = cuda_Piece[nChess];// �N�Ӧs�b�JnSrc!=0
            //pMove = cuda_KingMoves[nSrc][tid];
            pMove = tex1Dfetch(texture_KingMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x]=move;
        }
    }
    //��****************************************************************************
    else if( blockIdx.x==1 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+1;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_xRookMoves,x*512*12+cuda_xBitBoard[y]*12+threadIdx.x);
            nDst = (nSrc & 0xF0) | pMove;	// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+8]=move;
        }
    }
    else if( blockIdx.x==2 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+1;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_yRookMoves,y*1024*12+cuda_yBitBoard[x]*12+threadIdx.x);
            nDst = pMove | x;				// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+20]=move;
        }
    }
    else if( blockIdx.x==3 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+2;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_xRookMoves,x*512*12+cuda_xBitBoard[y]*12+threadIdx.x);
            nDst = (nSrc & 0xF0) | pMove;	// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+32]=move;
        }
    }
    else if( blockIdx.x==4 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+2;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_yRookMoves,y*1024*12+cuda_yBitBoard[x]*12+threadIdx.x);
            nDst = pMove | x;				// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+44]=move;
        }
    }    
    //��****************************************************************************
    else if( blockIdx.x==5 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+3;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_xCannonMoves,x*512*12+cuda_xBitBoard[y]*12+threadIdx.x);
            nDst = (nSrc & 0xF0) | pMove;	// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+56]=move;
        }
    }
    else if( blockIdx.x==6 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+3;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //�����a�V����
            pMove = tex1Dfetch(texture_yCannonMoves,y*1024*12+cuda_yBitBoard[x]*12+threadIdx.x);
            nDst = pMove | x;				// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+68]=move;
        }
    }
    else if( blockIdx.x==7 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+4;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_xCannonMoves,x*512*12+cuda_xBitBoard[y]*12+threadIdx.x);
            nDst = (nSrc & 0xF0) | pMove;	// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+80]=move;
        }
    }
    else if( blockIdx.x==8 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+4;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //�����a�V����
            pMove = tex1Dfetch(texture_yCannonMoves,y*1024*12+cuda_yBitBoard[x]*12+threadIdx.x);
            nDst = pMove | x;				// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+92]=move;
        }
    }
    //��****************************************************************************
    else if( blockIdx.x==9 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+5;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_KnightMoves,nSrc*12+threadIdx.x);
            nDst = pMove;
            if( !cuda_Board[nSrc+cuda_nHorseLegTab[nDst-nSrc+256]] )//�䰨�}
            {					
                WRITE_2_MOVE;
            }
            else
            {
                move=0;
            }
           cuda_move[threadIdx.x+104]=move;
        }
    }
    else if( blockIdx.x==10 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+6;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_KnightMoves,nSrc*12+threadIdx.x);
            nDst = pMove;
            if( !cuda_Board[nSrc+cuda_nHorseLegTab[nDst-nSrc+256]] )//�䰨�}
            {					
                WRITE_2_MOVE;
            }
            else
            {
                move=0;
            }
           cuda_move[threadIdx.x+116]=move;
        }
    }
    //�H****************************************************************************
    else if( blockIdx.x==11 )
    {
        if(threadIdx.x<8)
        {
            nChess=k+7;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_BishopMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            if( !cuda_Board[(nSrc+nDst)>>1] )//�H���L�l
            {
                WRITE_2_MOVE;
            }
            else
            {
                move=0;
            }
           cuda_move[threadIdx.x+128]=move;
        }
    }
    else if( blockIdx.x==12 )
    {
        if(threadIdx.x<8)
        {
            nChess=k+8;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_BishopMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            if( !cuda_Board[(nSrc+nDst)>>1] )//�H���L�l
            {
                WRITE_2_MOVE;
            }
            else
            {
                move=0;
            }
           cuda_move[threadIdx.x+136]=move;
        }
    }
    //�h****************************************************************************
    else if( blockIdx.x==13 )
    {
        if(threadIdx.x<8)
        {
            nChess=k+9;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_GuardMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+144]=move;
        }
    }
    else if( blockIdx.x==14 )
    {
        if(threadIdx.x<8)
        {
            nChess=k+10;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_GuardMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+152]=move;
        }
    }
    //�L****************************************************************************
    else if( blockIdx.x==15 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+11;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            //pMove = cuda_PawnMoves[Player][nSrc][(tid-160)%4];
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+160]=move;
        }
    }  
    else if( blockIdx.x==16 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+12;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            //pMove = cuda_PawnMoves[Player][nSrc][(tid-160)%4];
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+164]=move;
        }
    } 
    else if( blockIdx.x==17 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+13;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            //pMove = cuda_PawnMoves[Player][nSrc][(tid-160)%4];
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+168]=move;
        }
    } 
    else if( blockIdx.x==18 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+14;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            //pMove = cuda_PawnMoves[Player][nSrc][(tid-160)%4];
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+172]=move;
        }
    } 
    else if( blockIdx.x==19 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+15;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+176]=move;
        }
    } 
}

__global__ void cudaMoveGen(const unsigned int k,unsigned int* cuda_move)
{    
    //int tid=blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int  move, nSrc, nDst, x, y, nChess;
    unsigned char pMove;
    //�N��**************************************************************************
    if(blockIdx.x==0)
    {
        if(threadIdx.x<8)
        {
            nChess=k;
            nSrc = cuda_Piece[nChess];// �N�Ӧs�b�JnSrc!=0
            //pMove = cuda_KingMoves[nSrc][tid];
            pMove = tex1Dfetch(texture_KingMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x]=move;
        }
    }
    //��****************************************************************************
    else if( blockIdx.x==1 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+1;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_xRookMoves,x*512*12+cuda_xBitBoard[y]*12+threadIdx.x);
            nDst = (nSrc & 0xF0) | pMove;	// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+8]=move;
        }
    }
    else if( blockIdx.x==2 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+1;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_yRookMoves,y*1024*12+cuda_yBitBoard[x]*12+threadIdx.x);
            nDst = pMove | x;				// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+20]=move;
        }
    }
    else if( blockIdx.x==3 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+2;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_xRookMoves,x*512*12+cuda_xBitBoard[y]*12+threadIdx.x);
            nDst = (nSrc & 0xF0) | pMove;	// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+32]=move;
        }
    }
    else if( blockIdx.x==4 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+2;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_yRookMoves,y*1024*12+cuda_yBitBoard[x]*12+threadIdx.x);
            nDst = pMove | x;				// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+44]=move;
        }
    }    
    //��****************************************************************************
    else if( blockIdx.x==5 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+3;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_xCannonMoves,x*512*12+cuda_xBitBoard[y]*12+threadIdx.x);
            nDst = (nSrc & 0xF0) | pMove;	// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+56]=move;
        }
    }
    else if( blockIdx.x==6 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+3;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //�����a�V����
            pMove = tex1Dfetch(texture_yCannonMoves,y*1024*12+cuda_yBitBoard[x]*12+threadIdx.x);
            nDst = pMove | x;				// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+68]=move;
        }
    }
    else if( blockIdx.x==7 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+4;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //������V���ʡJ
            pMove = tex1Dfetch(texture_xCannonMoves,x*512*12+cuda_xBitBoard[y]*12+threadIdx.x);
            nDst = (nSrc & 0xF0) | pMove;	// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+80]=move;
        }
    }
    else if( blockIdx.x==8 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+4;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            x = nSrc & 0xF;// �Z4�즳��
            y = nSrc >> 4;// �e4�즳��
            //�����a�V����
            pMove = tex1Dfetch(texture_yCannonMoves,y*1024*12+cuda_yBitBoard[x]*12+threadIdx.x);
            nDst = pMove | x;				// 0x y|x  �e4��=y*16�A �Z4��=x
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+92]=move;
        }
    }
    //��****************************************************************************
    else if( blockIdx.x==9 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+5;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_KnightMoves,nSrc*12+threadIdx.x);
            nDst = pMove;
            if( !cuda_Board[nSrc+cuda_nHorseLegTab[nDst-nSrc+256]] )//�䰨�}
            {					
                WRITE_2_MOVE;
            }
            else
            {
                move=0;
            }
           cuda_move[threadIdx.x+104]=move;
        }
    }
    else if( blockIdx.x==10 )
    {
        if(threadIdx.x<12)
        {
            nChess=k+6;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_KnightMoves,nSrc*12+threadIdx.x);
            nDst = pMove;
            if( !cuda_Board[nSrc+cuda_nHorseLegTab[nDst-nSrc+256]] )//�䰨�}
            {					
                WRITE_2_MOVE;
            }
            else
            {
                move=0;
            }
           cuda_move[threadIdx.x+116]=move;
        }
    }
    //�H****************************************************************************
    else if( blockIdx.x==11 )
    {
        if(threadIdx.x<8)
        {
            nChess=k+7;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_BishopMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            if( !cuda_Board[(nSrc+nDst)>>1] )//�H���L�l
            {
                WRITE_2_MOVE;
            }
            else
            {
                move=0;
            }
           cuda_move[threadIdx.x+128]=move;
        }
    }
    else if( blockIdx.x==12 )
    {
        if(threadIdx.x<8)
        {
            nChess=k+8;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_BishopMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            if( !cuda_Board[(nSrc+nDst)>>1] )//�H���L�l
            {
                WRITE_2_MOVE;
            }
            else
            {
                move=0;
            }
           cuda_move[threadIdx.x+136]=move;
        }
    }
    //�h****************************************************************************
    else if( blockIdx.x==13 )
    {
        if(threadIdx.x<8)
        {
            nChess=k+9;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_GuardMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+144]=move;
        }
    }
    else if( blockIdx.x==14 )
    {
        if(threadIdx.x<8)
        {
            nChess=k+10;
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_GuardMoves,nSrc*8+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+152]=move;
        }
    }
    //�L****************************************************************************
    else if( blockIdx.x==15 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+11;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            //pMove = cuda_PawnMoves[Player][nSrc][(tid-160)%4];
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+160]=move;
        }
    }  
    else if( blockIdx.x==16 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+12;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            //pMove = cuda_PawnMoves[Player][nSrc][(tid-160)%4];
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+164]=move;
        }
    } 
    else if( blockIdx.x==17 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+13;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            //pMove = cuda_PawnMoves[Player][nSrc][(tid-160)%4];
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+168]=move;
        }
    } 
    else if( blockIdx.x==18 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+14;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            //pMove = cuda_PawnMoves[Player][nSrc][(tid-160)%4];
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+172]=move;
        }
    } 
    else if( blockIdx.x==19 )
    {
        if(threadIdx.x<4)
        {
            nChess=k+15;
            int Player;
            if(k<32){Player=0;}
            else{Player=1;}
            nSrc = cuda_Piece[nChess];// �Ѥl�s�b�JnSrc!=0
            pMove = tex1Dfetch(texture_PawnMoves,Player*256*4+nSrc*4+threadIdx.x);
            nDst = pMove;
            WRITE_2_MOVE;
           cuda_move[threadIdx.x+176]=move;
        }
    } 
}
__global__ void cuda_null()
{

}
//�I�scuda_MoveGen_null
void call_cudaMoveGen_null(const unsigned int nChess,int Board[256],int Piece[48],unsigned int xBitBoard[16],unsigned int yBitBoard[16],unsigned int * &ChessMove,unsigned short HistoryRecord[65535])
{
    cuda_null<<<20,numOfThreads>>>();
    hipDeviceSynchronize();
}
hipEvent_t start_timer, stop_timer; 
float time_timer;
//�I�scall_cudatimer
float call_cudatimer(int i)
{
    if(i)
    {
        hipEventRecord( stop_timer, 0 ); 
        hipEventSynchronize( stop_timer ); 
        hipEventElapsedTime( &time_timer, start_timer, stop_timer );
        hipEventDestroy( start_timer );
        hipEventDestroy( stop_timer );
        return time_timer;
    }
    else
    {
        hipEventCreate(&start_timer);
        hipEventCreate(&stop_timer); 
        hipEventRecord( start_timer, 0 ); 
        return 0.0;
    }
}
//�I�scuda_MoveGen
void call_cudaMoveGen(const unsigned int nChess,int Board[256],int Piece[48],unsigned int xBitBoard[16],unsigned int yBitBoard[16],unsigned int * &ChessMove,unsigned short HistoryRecord[65535])
{
    //copy Board[256]�MPiece[48] �ѽL��e���A
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cuda_Board),Board,1024));
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cuda_Piece),Piece,192));
    //copy  xBitBoard yBitBoard
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cuda_xBitBoard),xBitBoard,64));
    cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(cuda_yBitBoard),yBitBoard,64));

    //�ˬd�֤߹B��ɶ�
<<<<<<< .mine
    int testLoop=100000;
    hipEvent_t start1, stop1; 
    float time1;
    hipEventCreate(&start1);
    hipEventCreate(&stop1); 
    hipEventRecord( start1, 0 ); 
    for(int i=0;i<testLoop;i++)
    {
    cuda_null<<<1,numOfThreads>>>();
    hipDeviceSynchronize();
    }
    hipEventRecord( stop1, 0 ); 
    hipEventSynchronize( stop1 ); 
    hipEventElapsedTime( &time1, start1, stop1 );
    hipEventDestroy( start1 );
    hipEventDestroy( stop1 );

    hipEvent_t start2, stop2; 
    float time2;
    hipEventCreate(&start2);
    hipEventCreate(&stop2); 
    hipEventRecord( start2, 0 ); 
    for(int i=0;i<testLoop;i++)
    {
    cudaMoveGen<<<1,numOfThreads>>>(nChess,cuda_move);
    }
    hipDeviceSynchronize();
    hipEventRecord( stop2, 0 ); 
    hipEventSynchronize( stop2 ); 
    hipEventElapsedTime( &time2, start2, stop2 );
    hipEventDestroy( start2 );
    hipEventDestroy( stop2 );
    printf("time[gpu]: %g ms\n",time2);
=======
    //int testLoop=10000;
    //double t0=(double)clock()/CLOCKS_PER_SEC;
    //for(int i=0;i<testLoop;i++)
    //{
    //cuda_null<<<1,numOfThreads>>>();
    //hipDeviceSynchronize();
    //}
    //t0=((double)clock()/CLOCKS_PER_SEC-t0);
    //printf("time[null]: %g ms\n",t0*1000);
    //double t1=(double)clock()/CLOCKS_PER_SEC;
    //for(int i=0;i<testLoop;i++)
    //{
    cudaMoveGen<<<20,32>>>(nChess,cuda_move);
    //hipDeviceSynchronize();
    //}
    //t1=((double)clock()/CLOCKS_PER_SEC-t1);
    //printf("time[gpu]: %g ms\n",t1*1000);
    //printf("***warp*** pure time: %g ms\n",(t1-t0)*1000);
>>>>>>> .r15



    //�⵲�Gcopy�X��
    unsigned int host_move[numOfThreads];
    cutilSafeCall(hipMemcpy( host_move, cuda_move, 4*numOfThreads, hipMemcpyDeviceToHost)); 
    //�L�X������
    //for(int i=0;i<numOfThreads;i++)
    //{
    //    printf("thread %d -> move[%d] = %u\n",i,i,host_move[i]);
    //}

    ////������ܥd�O����
    //hipFree(cuda_move);

    for(int i=0;i<numOfThreads;i++)
    {
        if(host_move[i])
        {
            *(ChessMove++) = (HistoryRecord[host_move[i]]<<16) | host_move[i];
        }
    }
}

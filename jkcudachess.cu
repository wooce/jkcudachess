#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////////////////////////////////
// �����JBinghewusi.cpp                                                                                 //
// *******************************************************************************************************//
// ����H�ѳq�Τ���----�L�e���|�A����m����H�ѳq�Τ�����ĳ�n(Universal Chinese Chess Protocol�A²��ucci) //
// �@�̡J �S �w �x                                                                                        //
// ���J �����l���Ǭ�s�|                                                                            //
// �l�c�J fan_de_jun@sina.com.cn                                                                          //
//  QQ �J 83021504                                                                                        //
// *******************************************************************************************************//
// �\��J                                                                                                 //
// 1. ����x���ε{�����J�f�I                                                                              //
// 2. �z�Lucci��ĳ�P�ɭ��{�������i��q�T                                                                  //
////////////////////////////////////////////////////////////////////////////////////////////////////////////

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// includes, project
#include <cutil.h>
#include <cutil_inline.h>
// includes, kernels
#include <jkcudachess_kernel.cu>
// includes, Binghewusi
#include <string.h>
#include "ucci.h"
#include "FenBoard.h"
#include "Search.h"

void call_vecAdd();

int main(int argc, char* argv[])
{
	//call_vecAdd();
	int n;
	const char *BoolValue[2] = { "false", "true" };
	const char *ChessStyle[3] = { "solid", "normal", "risky" };
	char *BackSlashPtr;
	char BookFile[1024];
	UcciCommEnum IdleComm;
	UcciCommStruct Command;
	int ThisMove;
	
	printf("*******************************�L�e���| V0.60*********************************\n");
	printf("** �@�̡J�S�w�x                                                             **\n");
	printf("** ����m����H�ѳq�Τ�����ĳ�n(Universal Chinese Chess Protocol�A²��UCCI) **\n");
	printf("** �����i�H�Ρ��H����ElephantBoard�ɭ��{��(�@�̡J����)�[���C                **\n");
	printf("** �w��ϥΡ��L�e���| V0.60������H�Ѥ���                                   **\n");	
	printf("******************************************************************************\n");
	printf("����Jucci���O......\n");

	// ��������"ucci"���O
	if(BootLine() == UCCI_COMM_UCCI)
	{
		// �M������Ҧb���ؿ�argv[0]�A�åB��"BOOK.DAT"�q�{���ʬ٪��}���w�}���w
		BackSlashPtr = strrchr(argv[0], '\\');
		if (BackSlashPtr == 0) 
			strcpy(BookFile, "BOOK.DAT");
		else
		{
			strncpy(BookFile, argv[0], BackSlashPtr + 1 - argv[0]);
			strcpy(BookFile + (BackSlashPtr + 1 - argv[0]), "BOOK.DAT");
		}

		// �ե�CSearch���A�c�y��ƪ�l�Ƥ@�Ǭ����Ѽ�
		//a.��l�Ƶ۪k�w���ͼƲ�
		//b.��l��Hash��A���o21+1=22��Hash��A64M
		//c.�M�ž��v�ҵo��
		CSearch ThisSearch;

		// ��ܤ������W�١B�����B�@�̩M�ϥΪ�
		printf("\n");
		printf("id name ����H�Ѥ����L�e���|V0.60jk��\n");
		fflush(stdout);
		printf("id copyright ���v�Ҧ�(C) 2005-2008\n");
		fflush(stdout);
		printf("id author �S�w�x(�����l���Ǭ�s�|)\n");
		fflush(stdout);
		printf("id user CUDA\n\n");
		fflush(stdout);

		// ��ܤ���ucci���O���^�X�T���A��ܤ����Ҥ�����ﶵ
		// option batch %d
		printf("option batch type check default %s\n", BoolValue[ThisSearch.bBatch]);
		fflush(stdout);

		// option debug ��������X�ԲӪ��j���T���A�ëD�u�����ոռҦ��C
		printf("option debug type check default %s\n", BoolValue[ThisSearch.Debug]);
		fflush(stdout);

		// ���w�}���w��󪺦W�١A�i���w�h�Ӷ}���w���A�Τ�����;���j�}�A�p���������ϥζ}���w�A�i�H��ȳ]����
		ThisSearch.bUseOpeningBook = ThisSearch.m_Hash.LoadBook(BookFile);
		if(ThisSearch.bUseOpeningBook)
			printf("option bookfiles type string default %s\n", BookFile);
		else
			printf("option bookfiles type string default %s\n", 0);
		fflush(stdout);

		// �ݧ��w�W��
		printf("option egtbpaths type string default null\n");
		fflush(stdout);

		// ���Hash���j�p
		printf("option hashsize type spin default %d MB\n", ThisSearch.m_Hash.nHashSize*2*sizeof(CHashRecord)/1024/1024);
		fflush(stdout);

		// �������u�{��
		printf("option threads type spin default %d\n", 0);
		fflush(stdout);

		// �����F��۵M���۪��b�^�X��
		printf("option drawmoves type spin default %d\n", ThisSearch.NaturalBouts);
		fflush(stdout);

		// �ѳW
		printf("option repetition type spin default %d 1999�~���m����H���v�ɳW�h�n\n", UCCI_REPET_CHINESERULE);
		fflush(stdout);

		// �ŵ۵���O�_���}
		printf("option pruning type check %d\n", ThisSearch);
		fflush(stdout);

		// ���Ȩ�ƪ��ϥα��p
		printf("option knowledge type check %d\n", ThisSearch);
		fflush(stdout);

		// ���w��ܩʨt�ơA�q�`��0,1,2,3�|�ӯŧO�C�����Ȩ�ƥ[��@�w�d�򤺪��H���ơA�������C�����X���ۦP���ѡC
		printf("option selectivity type spin min 0 max 3 default %d\n", ThisSearch.nSelectivity);
		fflush(stdout);

		// ���w�U�Ѫ�����A�q�`��solid(�O�u)�Bnormal(����)�Mrisky(�_�i)�T��
		printf("option style type combo var solid var normal var risky default %s\n", ChessStyle[ThisSearch.nStyle]);
		fflush(stdout);		

		// copyprotection ��ܪ��v�ˬd�T��(���b�ˬd�A���v�T�����T�Ϊ��v�T�����~)�C 
		printf("copyprotection ok\n\n");
		fflush(stdout);

		// ucciok �o�Oucci���O���̫�@���^�X�T���A��ܤ����w�g�i�J��UCCI��ĳ�q�T�����A�C
		printf("ucciok\n\n");
		fflush(stdout);


		// �]�w�зǿ�X�M��l����
		ThisSearch.OutFile = stdout;	// �зǿ�X
		ThisSearch.fen.FenToBoard(Board, Piece, ThisSearch.Player, ThisSearch.nNonCapNum, ThisSearch.nCurrentStep, "rnbakabnr/9/1c5c1/p1p1p1p1p/9/9/P1P1P1P1P/1C5C1/9/RNBAKABNR r - - 0 1");
		ThisSearch.InitBitBoard(ThisSearch.Player, ThisSearch.nCurrentStep);
		printf("position fen rnbakabnr/9/1c5c1/p1p1p1p1p/9/9/P1P1P1P1P/1C5C1/9/RNBAKABNR r - - 0 1\n\n");
		fflush(stdout);
		

		// �}�l��������UCCI�R�O
		do 
		{
			IdleComm = IdleLine(Command, ThisSearch.Debug);
			switch (IdleComm) 
			{
				// isready �˴������O�_�B�_�N�����A�A��^�X�T���`�Oreadyok�A�ӫ��O�ȶȥΨ��˴������������O�����w�R�ϡ��O�_�ॿ�`�e�ǫ��O�C
				// readyok ��������B�_�N�����A(�Y�i�������O�����A)�A���ޤ����B�_�Ŷ����A�٬O��Ҫ��A�C
				case UCCI_COMM_ISREADY:
					printf("readyok\n");
					fflush(stdout);
					break;

				// stop ���_��������ҡA�j�s�X�ۡC�Z�x��ҨS���R���ɡA�N�θӫ��O�Ӥ����ҡA�M�᭫�s��J�����C
				case UCCI_COMM_STOP:
					ThisSearch.bStopThinking = 1;
					//printf("nobestmove\n");
					printf("score 0\n");
					fflush(stdout);
					break;

				// position fen �]�m�����m�ѽL���������A��fen�ӫ��wFEN�榡��Amoves�Z���򪺬O�H�Z���L���۪k
				case UCCI_COMM_POSITION:
					// �N�ɭ��ǨӪ�Fen����Ƭ��ѧ��T��
					ThisSearch.fen.FenToBoard(Board, Piece, ThisSearch.Player, ThisSearch.nNonCapNum, ThisSearch.nCurrentStep, Command.Position.szFenStr);
					ThisSearch.InitBitBoard(ThisSearch.Player, ThisSearch.nCurrentStep);

					// �N���������e�A�D�n�O���F��s�۪k�O���A�Τ_�`���˴��C
					for(n=0; n<Command.Position.nMoveNum; n++)
					{
						ThisMove = Move(Command.Position.lpdwCoordList[n]);
						if( !ThisMove )
							break;

						ThisSearch.MovePiece( ThisMove );
						ThisSearch.StepRecords[ThisSearch.nCurrentStep-1] |= ThisSearch.Checking(ThisSearch.Player) << 24;
					}

					ThisSearch.nBanMoveNum = 0;
					break;

				// banmoves ����e�����]�m�T��A�H�ѨM�����L�k�B�z���������D�C��X�{���������ɡA�Ѥ�i�H�ޱ��ɭ��V�����o�X�T����O�C
				case UCCI_COMM_BANMOVES:
					ThisSearch.nBanMoveNum = Command.BanMoves.nMoveNum;
					for(n=0; n<Command.BanMoves.nMoveNum; n++)
						ThisSearch.BanMoveList[n] = Move(Command.BanMoves.lpdwCoordList[n]);
					break;

				// setoption �]�m�����U�ذѼ�
				case UCCI_COMM_SETOPTION:
					switch(Command.Option.uoType) 
					{
						// setoption batch %d
						case UCCI_OPTION_BATCH:
							ThisSearch.bBatch = (Command.Option.Value.bCheck == TRUE);
							printf("option batch type check default %s\n", BoolValue[ThisSearch.bBatch]);
							fflush(stdout);
							break;

						// setoption debug %d ��������X�ԲӪ��j���T���A�ëD�u�����ոռҦ��C
						case UCCI_OPTION_DEBUG:
							ThisSearch.Debug = (Command.Option.Value.bCheck == TRUE);
							printf("option debug type check default %s\n", BoolValue[ThisSearch.Debug]);
							fflush(stdout);
							break;

						// setoption bookfiles %s  ���w�}���w��󪺦W�١A�i���w�h�Ӷ}���w���A�Τ�����;���j�}�A�p���������ϥζ}���w�A�i�H��ȳ]����
						case UCCI_OPTION_BOOKFILES:
							strcpy(BookFile, Command.Option.Value.szString);
							printf("option bookfiles type string default %s\n", BookFile);
							fflush(stdout);
							break;

						// setoption egtbpaths %s  ���w�ݧ��w��󪺦W�١A�i���w�h�Ӵݧ��w���|�A�Τ�����;���j�}�A�p���������ϥδݧ��w�A�i�H��ȳ]����
						//case e_OptionEgtbPaths:
							// �����ثe������}���w
							//printf("option egtbpaths type string default null\n");
							//fflush(stdout);
							//break;

						// setoption hashsize %d  �HMB�����W�wHash���j�p�A-1����������۰ʤ��oHash��C1��1024MB
						// �H���ɭ�����Bug�A�C���]�m�����ɡA�o�өR�O���b�}���w���e��
						case UCCI_OPTION_HASHSIZE:
							// -1MB(�۰�), 0MB(�۰�), 1MB(16), 2MB(17), 4MB(18), 8MB(19), 16MB(20), 32MB(21), 64MB(22), 128MB(23), 256MB(24), 512MB(25), 1024MB(26)
							if( Command.Option.Value.nSpin <= 0)
								n = 22;		// �ʬٱ��p�U�A�����۰ʤ��o(1<<22)*16=64MB�A���P�¨�U��A����U�@�b�C
							else
							{
								n = 15;											// 0.5 MB = 512 KB �H�������
								while( Command.Option.Value.nSpin > 0 )
								{
									Command.Option.Value.nSpin >>= 1;			// �C�����H2�A���쬰0
									n ++;
								}
							}								

							// ���[�J���s�˴����s�A�����۰ʤ��o�ɡAHash��j�p���i�Τ��s��1/2�C
							ThisSearch.m_Hash.DeleteHashTable();					// �����ϥ�delete���M���ª�Hash��
							ThisSearch.m_Hash.NewHashTable(n > 26 ? 26 : n, 12);	// ���������o�s��Hash��
							printf("option hashsize type spin default %d MB\n", ThisSearch.m_Hash.nHashSize*2*sizeof(CHashRecord)/1024/1024);	// ��ܹ�ڤ��o��Hash��j�p�A���JMB
							fflush(stdout);

							ThisSearch.m_Hash.ClearHashTable();
							ThisSearch.bUseOpeningBook = ThisSearch.m_Hash.LoadBook(BookFile);
							break;

						// setoption threads %d	      �������u�{�ơA���h�B�z���æ�B��A��
						case UCCI_OPTION_THREADS:
							// ThisSearch.nThreads = Command.Option.Value.Spin;		// 0(auto),1,2,4,8,16,32
							printf("option drawmoves type spin default %d\n", 0);
							fflush(stdout);
							break;

						// setoption drawmoves %d	  �F��۵M���۪��^�X��:50,60,70,80,90,100�A�H���w�g�۰���Ƭ��b�^�X��
						case UCCI_OPTION_DRAWMOVES:
							ThisSearch.NaturalBouts = Command.Option.Value.nSpin;
							printf("option drawmoves type spin default %d\n", ThisSearch.NaturalBouts);
							fflush(stdout);
							break;

						// setoption repetition %d	  �B�z�`�����ѳW�A�ثe�u���������H�ѴѳW1999��
						case UCCI_OPTION_REPETITION:
							// ThisSearch.nRepetitionStyle = Command.Option.Value.Repetition;
							// e_RepetitionAlwaysDraw  ���ܧ@�M
							// e_RepetitionCheckBan    �T����N
							// e_RepetitionAsianRule   �Ȭw�W�h
							// e_RepetitionChineseRule ����W�h�]�ʬ١^
							printf("option repetition type spin default %d", UCCI_REPET_CHINESERULE);
							printf("  ���L�e���|�������ثe���1999�~���m����H���v�ɳW�h�n\n");
							fflush(stdout);
							break;

						// setoption pruning %d�A���ŵۦV�e���š��O�_���}
						case UCCI_OPTION_PRUNING:
							ThisSearch.bPruning = Command.Option.Value.ugGrade;
							printf("option pruning type check %d\n", ThisSearch);
							fflush(stdout);
							break;

						// setoption knowledge %d�A���Ȩ�ƪ��ϥ�
						case UCCI_OPTION_KNOWLEDGE:
							ThisSearch.bKnowledge = Command.Option.Value.ugGrade;
							printf("option knowledge type check %d\n", ThisSearch);
							fflush(stdout);
							break;

						// setoption selectivity %d  ���w��ܩʨt�ơA�q�`��0,1,2,3�|�ӯŧO
						case UCCI_OPTION_SELECTIVITY:
							switch (Command.Option.Value.ugGrade)
							{
								case UCCI_GRADE_NONE:
									ThisSearch.SelectMask = 0;
									break;
								case UCCI_GRADE_SMALL:
									ThisSearch.SelectMask = 1;
									break;
								case UCCI_GRADE_MEDIUM:
									ThisSearch.SelectMask = 3;
									break;
								case UCCI_GRADE_LARGE:
									ThisSearch.SelectMask = 7;
									break;
								default:
									ThisSearch.SelectMask = 0;
									break;
							}
							printf("option selectivity type spin min 0 max 3 default %d\n", ThisSearch.SelectMask);
							fflush(stdout);
							break;

						// setoption style %d  ���w�U�Ѫ�����A�q�`��solid(�O�u)�Bnormal(����)�Mrisky(�_�i)�T��
						case UCCI_OPTION_STYLE:
							ThisSearch.nStyle = Command.Option.Value.usStyle;
							printf("option style type combo var solid var normal var risky default %s\n", ChessStyle[Command.Option.Value.usStyle]);
							fflush(stdout);
							break;						

						// setoption loadbook  UCCI�ɭ�ElephantBoard�b�C���s�شѧ��ɳ��|�o�e�o�����O
						case UCCI_OPTION_LOADBOOK:
							ThisSearch.m_Hash.ClearHashTable();
							ThisSearch.bUseOpeningBook = ThisSearch.m_Hash.LoadBook(BookFile);
							
							if(ThisSearch.bUseOpeningBook)
								printf("option loadbook succeed. %s\n", BookFile);		// ���\
							else
								printf("option loadbook failed! %s\n", "Not found file BOOK.DAT");				// �S���}���w
							fflush(stdout);
							printf("\n\n");
							fflush(stdout);
							break;

						default:
							break;
					}
					break;

				// Prepare timer strategy according to "go depth %d" or "go ponder depth %d" command
				case UCCI_COMM_GO:
				case UCCI_COMM_GOPONDER:
					switch (Command.Search.utMode)
					{
						// �T�w�`��
						case UCCI_TIME_DEPTH:
							ThisSearch.Ponder = 2;
							ThisSearch.MainSearch(Command.Search.DepthTime.nDepth);
							break;

						// �ɬq�s�J ���o�ɶ� = �Ѿl�ɶ� / �n�����B��
						case UCCI_TIME_MOVE:							
							ThisSearch.Ponder = (IdleComm == UCCI_COMM_GOPONDER ? 1 : 0);
							printf("%d\n", Command.Search.TimeMode.nMovesToGo);
							ThisSearch.MainSearch(127, Command.Search.DepthTime.nTime * 1000 / Command.Search.TimeMode.nMovesToGo, Command.Search.DepthTime.nTime * 1000);
							break;

						// �[�ɻs�J ���o�ɶ� = �C�B�W�[���ɶ� + �Ѿl�ɶ� / 20 (�Y���]�ѧ��|�b20�B������)
						case UCCI_TIME_INC:
							ThisSearch.Ponder = (IdleComm == UCCI_COMM_GOPONDER ? 1 : 0);
							ThisSearch.MainSearch(127, (Command.Search.DepthTime.nTime + Command.Search.TimeMode.nIncrement * 20) * 1000 / 20, Command.Search.DepthTime.nTime * 1000);
							break;

						default:
							break;
					}
					break;
			}
		} while (IdleComm != UCCI_COMM_QUIT);

		printf("bye\n");
		fflush(stdout);
	}

	return 0;
}

//�I�s�H���üư}�C�[�k����
void call_vecAdd()
{
	printf("test start");

	unsigned int num_threads = 4088;
	int  MAX_BLOCKTHREAD=512;
    unsigned int mem_size = sizeof( float) * num_threads;

	// allocate host memory
    float* h_idata1 = (float*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
        h_idata1[i] = (float) (rand()%100);
    }
	float* h_idata2 = (float*) malloc( mem_size);
    // initalize the memory
    for( unsigned int i = 0; i < num_threads; ++i) 
    {
        h_idata2[i] = (float) (rand()%100);
    }

    // allocate device memory
    float* d_idata1;
    cutilSafeCall( hipMalloc( (void**) &d_idata1, mem_size));
    // copy host memory to device
    cutilSafeCall( hipMemcpy( d_idata1, h_idata1, mem_size, hipMemcpyHostToDevice) );
    float* d_idata2;
    cutilSafeCall( hipMalloc( (void**) &d_idata2, mem_size));
    // copy host memory to device
    cutilSafeCall( hipMemcpy( d_idata2, h_idata2, mem_size, hipMemcpyHostToDevice) );

    // allocate device memory for result
    float* d_odata;
    cutilSafeCall( hipMalloc( (void**) &d_odata, mem_size));

    // execute the kernel
	dim3  grid( (( num_threads -1 ) / MAX_BLOCKTHREAD + 1) , 1);
    dim3  threads( MAX_BLOCKTHREAD , 1);
    vecAdd<<< grid, threads, mem_size >>>( d_idata1, d_idata2, d_odata);

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    // allocate mem for the result on host side
    float* h_odata = (float*) malloc( mem_size);
    // copy result from device to host
    cutilSafeCall( hipMemcpy( h_odata, d_odata, sizeof( float) * num_threads, hipMemcpyDeviceToHost) );

	//�L�X���G
	for( unsigned int i = 0; i < num_threads; ++i) 
    {
        printf("%f + %f = %f \n",h_idata1[i],h_idata2[i],h_odata[i]);
    }

	system("pause");
}